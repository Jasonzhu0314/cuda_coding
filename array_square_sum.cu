
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square_1(float *d_in, float *d_out) {
    int idx = threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f * f;
}



int main() {
    const int ARRAY_SIZE = 8;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);    

    // h前缀一般表示host， d前缀一般表示device
    float h_in[ARRAY_SIZE];
    for (int i = 0; i < ARRAY_SIZE; i++) {
        h_in[i] = float(i);
    }
    float h_out[ARRAY_SIZE];

    float* d_in;
    float* d_out;

    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_out, ARRAY_BYTES);

    // 从Host拷贝到Device
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    // 1个kernel，ARRAY_SIZE 线程
    square_1<<<1, ARRAY_SIZE>>>(d_in, d_out);

    // 从Device拷贝到Host
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    for (int i = 0; i < ARRAY_SIZE; i++) {
        printf("%f\n", h_out[i]);
    }
    
    // 释放内存
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}
